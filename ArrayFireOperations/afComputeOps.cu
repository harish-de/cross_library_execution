#include "hip/hip_runtime.h"
//
// Created by hkumar on 31.08.20.
//

#pragma once

#include <numeric>
#include "afComputeOps.cuh"

// begin of changes for unified interface

af::array afCompute::getAfFromCuda(int *device_data){
    cout << sizeof(device_data);
}

// end of changes for unified interface

af::array afCompute::getAFGpuData(std::vector<int> data) {

//    af::array deviceDummy;
//
//    auto start = high_resolution_clock::now();  // start time
//    for (int i = 0; i <= iteration; i++) {
    int* hostData = &data[0];
    af::array deviceData((dim_t)data.size(), hostData, afHost);
//    }
//    auto stop = high_resolution_clock::now(); // stop time
//    auto duration = duration_cast<nanoseconds>(stop - start); // time taken for performing the operation
//
//    std::cout << duration.count()/iteration << std::endl;

//    return deviceDummy;
    return deviceData;
}

af::array afCompute::afcpu2gpu(std::vector<int> data) {

    auto start = high_resolution_clock::now();  // start time
    for (int i = 0; i <= iteration; i++) {
        int* hostData = &data[0];
        af::array deviceData((dim_t)data.size(), hostData, afHost);
    }
    auto stop = high_resolution_clock::now(); // stop time
    auto duration = duration_cast<nanoseconds>(stop - start); // time taken for performing the operation

    std::cout << duration.count()/iteration << "\t";

    int* hostData = &data[0];
    af::array deviceData((dim_t)data.size(), hostData, afHost);
    return deviceData;
}

af::array afCompute::getAFGpuData(std::vector<float> data) {
    float * hostData = &data[0];
    af::array deviceData((dim_t)data.size(), hostData, afHost);
    return deviceData;
}

vector<int> afCompute::getAFCpuData(af::array deviceData) {
//    vector<int> hostDummy(deviceData.elements());
//    auto start = high_resolution_clock::now();  // start time
//    for (int i = 0; i <= iteration; i++) {
    vector<int> hostData(deviceData.elements());
    deviceData.host(hostData.data());
//    }
//    auto stop = high_resolution_clock::now(); // stop time
//    auto duration = duration_cast<nanoseconds>(stop - start); // time taken for performing the operation
//
//    std::cout << duration.count()/iteration << std::endl;
//    return hostDummy;
    return hostData;
}

vector<int> afCompute::afgpu2cpu(af::array deviceData) {
//    vector<int> hostDummy(deviceData.elements());
    auto start = high_resolution_clock::now();  // start time
    for (int i = 0; i <= iteration; i++) {
        vector<int> hostData(deviceData.elements());
        deviceData.host(hostData.data());
    }
    auto stop = high_resolution_clock::now(); // stop time
    auto duration = duration_cast<nanoseconds>(stop - start); // time taken for performing the operation

    std::cout << duration.count()/iteration << std::endl;
    vector<int> hostData(deviceData.elements());
    deviceData.host(hostData.data());
    return hostData;
}

vector<float> afCompute::getAFFloatCpuData(af::array deviceData) {
    vector<float> hostData(deviceData.elements());
    deviceData.host(hostData.data());
    return hostData;
}

af::array afCompute::afSelection(af::array deviceData, string operation, int value) {

    af::array device_result;

    if(!operation.compare("GE")) {
        auto start = high_resolution_clock::now();
        for (int i = 0; i <= iteration; i++) {
            device_result = af::where(af::operator>=(deviceData, value));
        }
        auto stop = high_resolution_clock::now(); // stop time
        auto duration = duration_cast<nanoseconds>(stop - start);

        std::cout << "selection: " << duration.count()/iteration << endl;
    }
    else if(!operation.compare("LE")) {
        auto start = high_resolution_clock::now();
        for (int i = 0; i <= iteration; i++) {
            device_result = af::where(af::operator<=(deviceData, value));
        }
        auto stop = high_resolution_clock::now(); // stop time
        auto duration = duration_cast<nanoseconds>(stop - start);

        std::cout << "selection: " << duration.count()/iteration << endl;
    }
    else if(!operation.compare("G")) {
        auto start = high_resolution_clock::now();
        for (int i = 0; i <= iteration; i++) {
            device_result = af::where(af::operator>(deviceData, value));
        }
        auto stop = high_resolution_clock::now(); // stop time
        auto duration = duration_cast<nanoseconds>(stop - start);

        std::cout << "selection: " << duration.count()/iteration << endl;
    }
    else if(!operation.compare("L")) {
        auto start = high_resolution_clock::now();
        for (int i = 0; i <= iteration; i++) {
            device_result = af::where(af::operator<(deviceData, value));
        }
        auto stop = high_resolution_clock::now(); // stop time
        auto duration = duration_cast<nanoseconds>(stop - start);

        std::cout << "selection: " << duration.count()/iteration << endl;
    }
    else if(!operation.compare("EQ")) {
        auto start = high_resolution_clock::now();
        for (int i = 0; i <= iteration; i++) {
            device_result = af::where(af::operator==(deviceData, value));
        }
        auto stop = high_resolution_clock::now(); // stop time
        auto duration = duration_cast<nanoseconds>(stop - start);

        std::cout << "selection: " << duration.count()/iteration << endl;
    }
    else{
        auto start = high_resolution_clock::now();
        for (int i = 0; i <= iteration; i++) {
            device_result = af::where(af::operator!=(deviceData, value));
        }
        auto stop = high_resolution_clock::now(); // stop time
        auto duration = duration_cast<nanoseconds>(stop - start);

        std::cout << "selection: " << duration.count()/iteration << endl;
    }

    return device_result;
}


af::array afCompute::afFilter(af::array deviceData, string operation, int value) {

    af::array device_result;

    auto start = high_resolution_clock::now();
    for (int i = 0; i <= iteration; i++) {
        device_result = af::where(af::operator<=(deviceData, value));
    }
    auto stop = high_resolution_clock::now(); // stop time
    auto duration = duration_cast<nanoseconds>(stop - start);
    cout << duration.count()/iteration << endl;

    return device_result;
}

af::array afCompute::afSelectionArrays(af::array lhs, string operation, af::array rhs) {

    af::array device_result;

    if(!operation.compare("GE")) {
        auto start = high_resolution_clock::now();
        for (int i = 0; i <= iteration; i++) {
            device_result = af::where(af::operator>=(lhs, rhs));
        }
        auto stop = high_resolution_clock::now(); // stop time
        auto duration = duration_cast<nanoseconds>(stop - start);
        std::cout << "selectionArrays: " << duration.count()/iteration << endl;
    }
    else if(!operation.compare("LE")) {
        auto start = high_resolution_clock::now();
        for (int i = 0; i <= iteration; i++) {
            device_result = af::where(af::operator<=(lhs, rhs));
        }
        auto stop = high_resolution_clock::now(); // stop time
        auto duration = duration_cast<nanoseconds>(stop - start);
        std::cout << "selectionArrays: " << duration.count()/iteration << endl;
    }
    else if(!operation.compare("G")) {
        auto start = high_resolution_clock::now();
        for (int i = 0; i <= iteration; i++) {
            device_result = af::where(af::operator>(lhs, rhs));
        }
        auto stop = high_resolution_clock::now(); // stop time
        auto duration = duration_cast<nanoseconds>(stop - start);
        std::cout << "selectionArrays: " << duration.count()/iteration << endl;
    }
    else if(!operation.compare("L")) {
        auto start = high_resolution_clock::now();
        for (int i = 0; i <= iteration; i++) {
            device_result = af::where(af::operator<(lhs, rhs));
        }
        auto stop = high_resolution_clock::now(); // stop time
        auto duration = duration_cast<nanoseconds>(stop - start);
        std::cout << "selectionArrays: " << duration.count()/iteration << endl;
    }
    else if(!operation.compare("EQ")) {
        auto start = high_resolution_clock::now();
        for (int i = 0; i <= iteration; i++) {
            device_result = af::where(af::operator==(lhs, rhs));
        }
        auto stop = high_resolution_clock::now(); // stop time
        auto duration = duration_cast<nanoseconds>(stop - start);
        std::cout << "selectionArrays: " << duration.count()/iteration << endl;
    }
    else{
        auto start = high_resolution_clock::now();
        for (int i = 0; i <= iteration; i++) {
            device_result = af::where(af::operator!=(lhs, rhs));
        }
        auto stop = high_resolution_clock::now(); // stop time
        auto duration = duration_cast<nanoseconds>(stop - start);
        std::cout << "selectionArrays: " << duration.count()/iteration << endl;
    }

    return device_result;
}

af::array afCompute::afConjunction(af::array deviceLHS, af::array deviceRHS) {
    auto start = high_resolution_clock::now();
    for (int i = 0; i <= iteration; i++) {
        af::array result = af::setIntersect(deviceLHS, deviceRHS, true);
    }
    auto stop = high_resolution_clock::now(); // stop time
    auto duration = duration_cast<nanoseconds>(stop - start);
    std::cout << "conjunction: " << duration.count()/iteration << endl;

    af::array result = af::setIntersect(deviceLHS, deviceRHS, true);
    return result;
}

af::array afCompute::afProduct(af::array deviceLHS, af::array deviceRHS) {
    auto start = high_resolution_clock::now();
    for (int i = 0; i <= iteration; i++) {
        af::array result = af::operator*(deviceLHS,deviceRHS);
    }
    auto stop = high_resolution_clock::now(); // stop time
    auto duration = duration_cast<nanoseconds>(stop - start);
    std::cout << "product: " << duration.count()/iteration << endl;

    af::array result = af::operator*(deviceLHS,deviceRHS);
    return result;
}

int afCompute::afSum(af::array deviceData) {
    auto start = high_resolution_clock::now();
    for (int i = 0; i <= iteration; i++) {
        int result = af::sum<int>(deviceData,(dim_t)0);
    }
    auto stop = high_resolution_clock::now(); // stop time
    auto duration = duration_cast<nanoseconds>(stop - start);
    std::cout << "sum: " << duration.count()/iteration << endl;

    int result = af::sum<int>(deviceData,(dim_t)0);
    return result;
}

float afCompute::afAvg(af::array deviceData) {
    float sum = af::sum<float>(deviceData,(dim_t)0);
    float result = sum/deviceData.elements();
    return result;
}

int afCompute::afCountIf(af::array deviceData,int value) {
//        return af::count<int>(deviceData);
    af::array index = af::where(af::operator==(deviceData,value));
    return af::count<int>(index);
}

int afCompute::afCount(af::array deviceData) {
    return deviceData.elements();
}


af::array test_nested_loop(af::array::array_proxy result,af::array::array_proxy child,af::array parent) { //

    for(int j=0 ; j < parent.elements(); j++){
        result = af::select(child==parent(j),j,result);
    }
    return result;
}


//af::array nested_loop_join(af::array::array_proxy child,
//                           af::array parent,
//                           af::array::array_proxy result) {
//
//    for (int j = 0; j < parent.elements(); j++){
//        result = af::select(child==parent(j),j,result);
//    }
//    return result;
//}

af::array afCompute::afJoin(af::array parent, af::array child) {

// Why arrayfire join is slow:
// https://stackoverflow.com/questions/50242141/arrayfire-cuda-application-is-extremely-slow-in-the-first-minute
// https://github.com/arrayfire/arrayfire-python/issues/140

/*
 *          condition = (child(i) == parent(j));
            A(i) = (condition)*j + (!condition)*A(i);
 */
    af::array A = af::constant(-1,child.elements());

    gfor(af::seq i, child.elements()) { //
        test_nested_loop(A(i),child(i),parent);
    }
//
    A = A.as(af::dtype::s32);

    return A;
}

// Please note arrayfire does not need prefix sum operation as it already returns the indices in selection operation
// the function has been added below for consistency to run all libraries in sequence
af::array afCompute::afPrefixSum(af::array deviceSelData) {
    return deviceSelData;
}

int afCompute::afFindMax(af::array deviceData) {
    return af::max<int>(deviceData);
}

int afCompute::afFindMin(af::array deviceData) {
    return af::min<int>(deviceData);
}

af::array afCompute::afSort(af::array deviceData, int order) {

    af::array sorted_data;
    auto start = high_resolution_clock::now();
    for (int i = 0; i <= iteration; i++) {
        if(order){
            sorted_data = af::sort(deviceData,0, false);
        }else{
            sorted_data = af::sort(deviceData);
        }
    }
    auto stop = high_resolution_clock::now(); // stop time
    auto duration = duration_cast<nanoseconds>(stop - start);
    std::cout << "sort: " << duration.count()/iteration << endl;
    return sorted_data;
}

af::array afCompute::afGroupBy(af::array keys, af::array values) {
    af::array keys_out;
    af::array values_out;
    auto start = high_resolution_clock::now();
    for (int i = 0; i <= iteration; i++) {
        af::sumByKey(keys_out, values_out, keys, values);
    }
    auto stop = high_resolution_clock::now(); // stop time
    auto duration = duration_cast<nanoseconds>(stop - start);
    cout << "sumByKey: " << duration.count()/iteration << endl;
//    af::sumByKey(keys_out, values_out, keys, values);
    return values_out;
}

af::array afCompute::afCountByKey(af::array data){
    af::array values_out;

//    auto start1 = high_resolution_clock::now();
//    for (int i = 0; i <= iteration; i++) {
//        data = afSort(data,0);
//    }
//    auto stop1 = high_resolution_clock::now(); // stop time
//    auto duration1 = duration_cast<nanoseconds>(stop1 - start1);

    auto start = high_resolution_clock::now();
    for (int i = 0; i <= iteration; i++) {
        // start time
        af::array keys_out;
        af::array temp = data;

        af::sumByKey(keys_out,values_out,data,temp);
        values_out = af::operator/(values_out,keys_out);
    }
    auto stop = high_resolution_clock::now(); // stop time
    auto duration = duration_cast<nanoseconds>(stop - start);

    cout << "countByKey: " << duration.count()/iteration << endl;

    return values_out;
}

af::array afCompute::afSumOfVectors(af::array vec1, af::array vec2) {
    af::array result(vec1.elements());

    auto start = high_resolution_clock::now();
    for (int i = 0; i <= iteration; i++) {
        gfor(af::seq i, vec1.elements()){
            result(i) = vec1(i) + vec2(i);
        }

        result = result.as(af::dtype::s32);
    }

    auto stop = high_resolution_clock::now(); // stop time
    auto duration = duration_cast<nanoseconds>(stop - start);

    std::cout << "grouping: " << duration.count()/iteration << endl;

    return result;
}

af::array afCompute::afSortByKey(af::array data, af::array dependent_data, int order) {

    af::array sorted_index;
    af::array sorted_value;

    auto start = high_resolution_clock::now();
    for (int i = 0; i <= iteration; i++) {
        if (order) {
            af::sort(sorted_index, sorted_value, data, dependent_data,0,false);
        }else{
            af::sort(sorted_index, sorted_value, data, dependent_data);
        }
    }

    auto stop = high_resolution_clock::now(); // stop time
    auto duration = duration_cast<nanoseconds>(stop - start);

    std::cout << "sortbykey: " << duration.count()/iteration << endl;
    return sorted_value;
}

af::array afCompute::afAvgByKey(af::array keys, af::array values) {
    auto start = high_resolution_clock::now();
    for (int i = 0; i <= iteration; i++) {
        af::array sums = afGroupBy(keys,values);

        af::array counts = afCountByKey(keys);

        af::array average = af::operator/(sums,counts);
    }

    auto stop = high_resolution_clock::now(); // stop time
    auto duration = duration_cast<nanoseconds>(stop - start);

    std::cout << "avgByKey: " << duration.count()/iteration << endl;

    af::array sums = afGroupBy(keys,values);

    af::array counts = afCountByKey(keys);

    af::array average = af::operator/(sums,counts);
    return average;
}